#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <math.h>
#include <time.h>
#include <chrono>
#include <iomanip>
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#define _USE_MATH_DEFINES
#define MAX_MOVES 32
#define MINIMAX_MAXVAL 50000
#define MINIMAX_MINVAL -50000
#define MASK 0xFFFFFFFF

using namespace std;
using namespace std::chrono;


//---------------------------------------------Funkcje GPU------------------------------------------------------------------
//-------------Szukanie pionkow gracza na planszy-----------------
__global__ void kernelFindPawns(int* board, int* pawns_id, bool whose_move)
{
	if(threadIdx.x < 64) {
		int pawn_pos = 1000;

		if(whose_move == 0 && board[threadIdx.x] > 0 && board[threadIdx.x] < 10)
			pawn_pos = threadIdx.x;
		else if(whose_move == 1 && board[threadIdx.x] > 10)
			pawn_pos = threadIdx.x;
		
		pawns_id[threadIdx.x] = pawn_pos;
	}
}

//-------------Wyznaczanie mozliwych ruchow-----------------
__inline__ __device__ 
bool checkMovePawn(int* board, int start_pos, int end_pos, bool whose_move)
{
	int end_field = board[end_pos];
	int start_row = start_pos / 8;
	int start_col = start_pos % 8;
	int end_row = end_pos / 8;
	int end_col = end_pos % 8;
	bool move_possible = 0;
	
	if(whose_move == 0){
		if(start_col == end_col) {
			if(end_row == start_row - 1 && end_field == 0)
				 move_possible = 1;
			else if(end_row == start_row - 2 && end_field == 0 && start_row == 6) {
				if(board[(start_row - 1)*8 + end_col] == 0)
					move_possible = 1;
			}			 
		}
		else if(abs(start_col - end_col) == 1) {
			if(end_row == start_row - 1 && end_field != 0)
				 move_possible = 1;
		}
	}
	else{
		if(start_col == end_col) {
			if(end_row == start_row + 1 && end_field == 0)
				 move_possible = 1;
			else if(end_row == start_row + 2 && end_field == 0 && start_row == 1) {
				if(board[(start_row + 1)*8 + end_col] == 0)
					move_possible = 1;
			}			 
		}
		else if(abs(start_col - end_col) == 1) {
			if(end_row == start_row + 1 && end_field != 0)
				 move_possible = 1;
		}
	}

	return move_possible;
}

__inline__ __device__ 
bool checkMoveKnight(int* board, int start_pos, int end_pos)
{
	int start_row = start_pos / 8;
	int start_col = start_pos % 8;
	int end_row = end_pos / 8;
	int end_col = end_pos % 8;
	bool move_possible = 0;
	
	if(abs(start_col - end_col) == 2 && abs(start_row - end_row) == 1)
		move_possible = 1;
	else if(abs(start_col - end_col) == 1 && abs(start_row - end_row) == 2)
		move_possible = 1;

	return move_possible;
}

__inline__ __device__ 
bool checkMoveRook(int* board, int start_pos, int end_pos)
{
	int start_row = start_pos / 8;
	int start_col = start_pos % 8;
	int end_row = end_pos / 8;
	int end_col = end_pos % 8;
	bool move_possible = 0;
	
	if(end_col == start_col) {
		move_possible = 1;
		int delta_row = end_row - start_row;
		int row_mult = 1;
		if(delta_row < 0)
			row_mult = -1;
		else
			delta_row++;

		for(int i = 1; i < abs(delta_row); i++) {
			int id = (start_row + i * row_mult) * 8 + end_col;
			if(board[id] != 0) {
				move_possible = 0;
				return move_possible;
			}
		}
	}
	else if(end_row == start_row) {
		move_possible = 1;
		int delta_col = end_col - start_col;
		int col_mult = 1;
		if(delta_col < 0)
			col_mult = -1;
		else
			delta_col++;

		for(int i = 1; i < abs(delta_col); i++) {
			int id = start_row * 8 + start_col + i * col_mult;
			if(board[id] != 0) {
				move_possible = 0;
				return move_possible;
			}
		}
	}

	return move_possible;
}

__inline__ __device__ 
bool checkMoveBishop(int* board, int start_pos, int end_pos)
{
	int start_row = start_pos / 8;
	int start_col = start_pos % 8;
	int end_row = end_pos / 8;
	int end_col = end_pos % 8;
	int delta_row = end_row - start_row;
	int delta_col = end_col - start_col;
	bool move_possible = 0;

	if(abs(delta_row) == abs(delta_col)) {
		move_possible = 1;
		int row_mult = 1;
		int col_mult = 1;
		if(delta_row < 0)
			row_mult = -1;
		if(delta_col < 0)
			col_mult = -1;

		for(int i = 1; i < abs(delta_row); i++) {
			int id = (start_row + i * row_mult) * 8 + start_col + i * col_mult;
			if(board[id] != 0) {
				move_possible = 0;
				return move_possible;
			}
		}
		
	}

	return move_possible;
}

__inline__ __device__ 
bool checkMoveQueen(int* board, int start_pos, int end_pos)
{
	int start_row = start_pos / 8;
	int start_col = start_pos % 8;
	int end_row = end_pos / 8;
	int end_col = end_pos % 8;
	bool move_possible = 0;

	if(start_col == end_col || start_row == end_row)
		move_possible = checkMoveRook(board, start_pos, end_pos);
	else if(abs(start_row - end_row) == abs(start_col - end_col))
		move_possible = checkMoveBishop(board, start_pos, end_pos);

	return move_possible;
}

__inline__ __device__ 
bool checkMoveKing(int* board, int start_pos, int end_pos)
{
	int start_row = start_pos / 8;
	int start_col = start_pos % 8;
	int end_row = end_pos / 8;
	int end_col = end_pos % 8;
	bool move_possible = 0;
	
	if(abs(start_col - end_col) == 1 ||abs(start_col - end_col) == 0) {
		if(abs(start_row - end_row) == 1 || abs(start_row - end_row) == 0)
			move_possible = 1;
	}

	return move_possible;
}


__global__ void kernelCheckAllMoves(int* board, int* moves, int start_id, bool whose_move)
{
	int pawn = board[start_id] % 10;
	if(threadIdx.x < 64) {
		int end_id = threadIdx.x;
		bool move_possible;

		if(whose_move == 0 && board[end_id] > 0 && board[end_id] < 10)
			move_possible = 0;
		else if(whose_move == 1 && board[end_id] > 10)
			move_possible = 0;
		else {
			if(pawn == 1)
				move_possible = checkMovePawn(board, start_id, end_id, whose_move);
			else if(pawn == 2)
				move_possible = checkMoveRook(board, start_id, end_id);
			else if(pawn == 3)
				move_possible = checkMoveBishop(board, start_id, end_id);
			else if(pawn == 4)
				move_possible = checkMoveKnight(board, start_id, end_id);
			else if(pawn == 5)
				move_possible = checkMoveQueen(board, start_id, end_id);
			else if(pawn == 6)
				move_possible = checkMoveKing(board, start_id, end_id);
		}

		int ret_val = 1000;
		if(move_possible == 1)
			ret_val = threadIdx.x;
		moves[threadIdx.x] = ret_val;
	}
}

__global__ void kernelCheckMove(int* board, int* move, int start_id, int end_id, bool whose_move)
{
	int pawn = board[start_id] % 10;
	if(threadIdx.x == 0) {
		bool move_possible;

		if(whose_move == 0 && board[end_id] > 0 && board[end_id] < 10)
			move_possible = 0;
		else if(whose_move == 1 && board[end_id] > 10)
			move_possible = 0;
		else {
			if(pawn == 1)
				move_possible = checkMovePawn(board, start_id, end_id, whose_move);
			else if(pawn == 2)
				move_possible = checkMoveRook(board, start_id, end_id);
			else if(pawn == 3)
				move_possible = checkMoveBishop(board, start_id, end_id);
			else if(pawn == 4)
				move_possible = checkMoveKnight(board, start_id, end_id);
			else if(pawn == 5)
				move_possible = checkMoveQueen(board, start_id, end_id);
			else if(pawn == 6)
				move_possible = checkMoveKing(board, start_id, end_id);
		}
		int ret_val = 1000;
		if(move_possible == 1)
			ret_val = threadIdx.x;
		move[threadIdx.x] = ret_val;
	}
}


//--------------------Obliczanie punktow planszy-------------------
__inline__ __device__ 
int getPointsPawn(int pos, int field_type, bool whose_move)
{
	int row = pos / 8;
	int col = pos % 8;
	int points = 10;

	if(col == 0 || col == 7)
		pos -= 2;
	
	if(field_type < 10) {
		if(row == 1)
			points += 30;
		else if(row == 0)
			points += 70;
	}
	else {
		if(row == 6)
			points += 30;
		else if(row == 7)
			points += 70;
	}

	if(whose_move == 0 && field_type > 10)
		points *= -1;
	else if(whose_move == 1 && field_type < 10)
		points *= -1;

	return points;
}

__inline__ __device__ 
int getPointsBishop(int pos, int field_type, bool whose_move)
{
	int row = pos / 8;
	int col = pos % 8;
	int points = 30;

	if(row == 0 || row == 7)
		points -= 4;
	else if(row >= 2 && row <= 5)
		points += 4;

	if(col == 0 || col == 7)
		points -= 4;
	else if(col >= 2 && col <= 5)
		points += 4;
	
	if(whose_move == 0 && field_type > 10)
		points *= -1;
	else if(whose_move == 1 && field_type < 10)
		points *= -1;

	return points;
}

__inline__ __device__ 
int getPointsKnight(int pos, int field_type, bool whose_move)
{
	int row = pos / 8;
	int col = pos % 8;
	int points = 30;

	if(whose_move == 0 && row == 0)
		points -= 8;
	else if(whose_move == 0 && row == 7)
		points -= 8;
	else if(row == 0 || row == 7)
		points -= 4;
	else if(row >= 2 && row <= 5)
		points += 4;

	if(col == 0 || col == 7)
		points -= 4;
	else if(col >= 2 && col <= 5)
		points += 4;
	
	if(whose_move == 0 && field_type > 10)
		points *= -1;
	else if(whose_move == 1 && field_type < 10)
		points *= -1;

	return points;
}

__inline__ __device__ 
int getPointsRook(int pos, int field_type, bool whose_move)
{
	//int row = pos / 8;
	int col = pos % 8;
	int points = 50;

	if(col == 0 || col == 7)
		pos -= 6;
	else if(col == 1 || col == 6)
		pos -= 3;
	else if(col == 3 || col == 4)
		pos += 3;

	if(whose_move == 0 && field_type > 10)
		points *= -1;
	else if(whose_move == 1 && field_type < 10)
		points *= -1;

	return points;
}

__inline__ __device__ 
int getPointsQueen(int pos, int field_type, bool whose_move)
{
	//int row = pos / 8;
	int col = pos % 8;
	int points = 90;

	if(col == 0 || col == 7)
		points -= 5;
	
	if(whose_move == 0 && field_type > 10)
		points *= -1;
	else if(whose_move == 1 && field_type < 10)
		points *= -1;

	return points;
}

__inline__ __device__ 
int getPointsKing(int pos, int field_type, bool whose_move)
{
	int row = pos / 8;
	int col = pos % 8;
	int points = 1500;

	if(row > 1 && row < 6)
		points -= 20;
	else if(row == 1 || row == 6)
		points -= 5;

	if(col == 3 || col == 4)
		points -= 5;

	if(whose_move == 0 && field_type > 10)
		points *= -1;
	else if(whose_move == 1 && field_type < 10)
		points *= -1;

	return points;
}


__inline__ __device__
int warpReductionPoints(int value) {
  for (int offset = warpSize/2; offset > 0; offset /= 2) 
    value += __shfl_down_sync(MASK, value, offset);

  return value;
}

__inline__ __device__ 
int blockReductionPoints(int value)
{
	static __shared__ double shared[32];
	int w_id = threadIdx.x / warpSize;
	int t_id = threadIdx.x % warpSize;

	value = warpReductionPoints(value);

	if(t_id == 0)
		shared[w_id] = value;

	__syncthreads();

	value = (threadIdx.x < blockDim.x / warpSize) ? shared[t_id] : 0;

	//Finalna redukcja w pierwszym warpie
	if(w_id == 0)
		value = warpReductionPoints(value);

	return value;
}

__global__ void kernelEvalPoints(const int* board, int* points_out, bool whose_move)
{
	if(threadIdx.x < 64) {
		int pos_id = threadIdx.x;
		int field = board[pos_id];
		int points = 0;

		if(field == 0)
			points = 0;
		else if(field % 10 == 1) 
			points = getPointsPawn(pos_id, field, whose_move);
		else if(field % 10 == 2)
			points = getPointsRook(pos_id, field, whose_move);
		else if(field % 10 == 3) 
			points = getPointsBishop(pos_id, field, whose_move);
		else if(field % 10 == 4) 
			points = getPointsKnight(pos_id, field, whose_move);
		else if(field % 10 == 5) 
			points = getPointsQueen(pos_id, field, whose_move);
		else if(field % 10 == 6) 
			points = getPointsKing(pos_id, field, whose_move);

		points = blockReductionPoints(points);
		if(threadIdx.x == 0)
			points_out[blockIdx.x] = points;
	}
}

//----------------------------Minimax-------------------------------------------
__global__ void kernelMax(int* max, int* points, unsigned int rozmiar)
{
	__shared__ int max_values[MAX_MOVES];
    int t_id = threadIdx.x;
	unsigned int b_id = blockIdx.x * blockDim.x + threadIdx.x;

	max_values[t_id] = points[b_id];
	__syncthreads();

	for (int i = 1; i < MAX_MOVES; i *= 2) {
		if((t_id + i) < MAX_MOVES && max_values[t_id + i] != MINIMAX_MAXVAL) {
			if (max_values[t_id + i] > max_values[t_id])
				max_values[t_id] = max_values[t_id + i];
		}
		__syncthreads();
	}
			
	if (t_id == 0)
		max[blockIdx.x] = max_values[t_id];
}

__global__ void kernelMin(int* min, int* points, unsigned int rozmiar)
{
	__shared__ int min_values[MAX_MOVES];
    int t_id = threadIdx.x;
	unsigned int b_id = blockIdx.x * blockDim.x + threadIdx.x;

	min_values[t_id] = points[b_id];
	__syncthreads();

	for (int i = 1; i < MAX_MOVES; i *= 2) {
		if((t_id + i) < MAX_MOVES) {
			if (min_values[t_id + i] < min_values[t_id] && min_values[t_id + i] != MINIMAX_MINVAL)
				min_values[t_id] = min_values[t_id + i];
		}
		__syncthreads();
	}
	
	if (t_id == 0)
		min[blockIdx.x] = min_values[t_id];	
}


//------------------------------------------------Funkcje globalne----------------------------------------------------------
void resetuj_plansze(int** plansza, int* gracze) 
{
	/*Oznaczenia pionkow:
	0 - puste pole
	1/11 - pion gracza bialego/czarnego
	2/12 - wieza gracza bialego/czarnego
	3/13 - goniec gracza bialego/czarnego
	4/14 - skoczek gracza bialego/czarnego
	5/15 - hetman gracza bialego/czarnego
	6/16 - krol gracza bialego/czarnego
	*/
	for(int i = 0; i < 8; i++) {
		for(int j = 0; j < 8; j++) {
			int id = 0;
			//Dwa gorne rzedy na pionki gracza czarnego
			if(i == 0) {
				if(j == 0 || j == 7)
					id = 12;
				else if(j == 1 || j == 6)
					id = 14;
				else if(j == 2 || j == 5)
					id = 13;
				else if(j == 3)
					id = 15;
				else if(j == 4)
					id = 16;
				
			}
			else if(i == 1) {
				id = 11;
			}
			//Dwa dolne rzedy na pionki gracza bialego
			else if(i == 7) {
				if(j == 0 || j == 7)
					id = 2;
				else if(j == 1 || j == 6)
					id = 4;
				else if(j == 2 || j == 5)
					id = 3;
				else if(j == 3)
					id = 5;
				else if(j == 4)
					id = 6;
				
			}
			else if(i == 6) {
				id = 1;
			}
			//Reszta pol bez pionkow

			plansza[i][j] = id;
		}	
	}
	while(true) {
		system("clear");
		int wartosc = -1;
		cout << "Podaj gracza białego (0 - człowiek, 1 - AI)" << endl;
		cin >> wartosc;
		if(wartosc == 0 || wartosc == 1) {
			gracze[0] = wartosc;
			break;
		}
	}
	while(true) {
		system("clear");
		int wartosc = -1;
		cout << "Podaj gracza czarnego (0 - człowiek, 1 - AI)" << endl;
		cin >> wartosc;
		if(wartosc == 0 || wartosc == 1) {
			gracze[1] = wartosc;
			break;
		}
	}
}

void rysuj_plansze(int** plansza) 
{
	cout << "=||";
	for(int i = 0; i < 8; i++) {
		cout << "===|";
	}
	cout << "|";
	cout << endl;
	for(int i = 0; i < 8; i++) {
		cout << 8 - i;
		cout << "||";
		for(int j = 0; j < 8; j++) {
			int id = plansza[i][j];
			char rysuj = ' ';
			
			if(id % 10 == 1) {
				if(id < 10)
					rysuj = 'P';
				else
					rysuj = 'p';
			}
			else if(id % 10 == 2) {
				if(id < 10)
					rysuj = 'R';
				else
					rysuj = 'r';
			}
			else if(id % 10 == 3) {
				if(id < 10)
					rysuj = 'B';
				else
					rysuj = 'b';
			}
			else if(id % 10 == 4) {
				if(id < 10)
					rysuj = 'N';
				else
					rysuj = 'n';
			}
			else if(id % 10 == 5) {
				if(id < 10)
					rysuj = 'Q';
				else
					rysuj = 'q';
			}
			else if(id % 10 == 6) {
				if(id < 10)
					rysuj = 'K';
				else
					rysuj = 'k';
			}
			
			cout << " " << rysuj << " " << "|";
		}
		cout << "|";

		//Wypisywanie legendy
		switch(i) {
			case 0:
				cout << "	Oznaczenia:";
				break;
			case 1:
				cout << "	male litery - pionki czarne";
				break;
			case 2:
				cout << "	R - wieza";
				break;
			case 3:
				cout << "	N - skoczek";
				break;
			case 4:
				cout << "	K - krol";
				break;
		}

		cout << endl;
		if(i < 7) {
			cout << "-++";
			for(int k = 0; k < 8; k++) {
				cout << "---+";
			}
			cout << "+";
		}
		else {
			cout << "=++";
			for(int k = 0; k < 8; k++) {
				cout << "===+";
			}
			cout << "+";
		}
		
		switch(i) {
			case 0:
				cout << "	DUZE LITERY - pionki biale";
				break;
			case 1:
				cout << "	P - pion";
				break;
			case 2:
				cout << "	B - goniec";
				break;
			case 3:
				cout << "	Q - hetman";
				break;
		}
		cout << endl;
	}
	
	cout << " ||";
	for(int i = 0; i < 8; i++) {
		cout << " " << char('A' + i) << " " << "|";
	}
	cout << "|";
	cout << endl << endl;
}

//Zamiana 2 pierwszych znakow wczytanego od gracza stringa na indeksy rzedow i kolumn tablicy
//Jesli dowolny indeks wykracza poza tablice(indeks = <0, 7>) zwracamy wartosc false 
bool string_na_pole(int** plansza, string pole_str, int* pole)
{
	int kolumna = int(tolower(pole_str[0])) - (int)'a';
	int rzad = (int)'8' - (int)pole_str[1];

	if(rzad >= 0 && rzad <= 7)
		pole[0] = rzad;
	else
		return 0;
		
	if(kolumna >= 0 && kolumna <= 7)
		pole[1] = kolumna;
	else
		return 0;	
	
	return 1;
}

string pole_na_string(int** plansza, int* pole)
{
	string pole_str;
	pole_str[0] = char(pole[1] + 65);
	pole_str[1] = char(56 - pole[0]);

	return pole_str;
}

//Zwraca typ pionka (0 jesli pionek jest nieprawidlowy)
int sprawdz_pionek(int** plansza, int* pole, bool czyj_ruch) 
{
	int kolumna = pole[1];
	int rzad = pole[0];
	
	int pionek = plansza[rzad][kolumna];
	if(pionek == 0)
		return 0;
	else if(pionek > 0 && pionek < 10 && czyj_ruch == 0)
		return pionek % 10;
	else if(pionek > 10 && czyj_ruch == 1)
		return pionek % 10;
	
	return 0;
}

//Zwraca nazwe pionka na wybranym polu
string nazwa_pola(int** plansza, int* pole) {
	int pole_doc_typ = plansza[pole[0]][pole[1]];
	if(pole_doc_typ == 0)
		return "Puste pole";
	else if(pole_doc_typ % 10 == 1)
		return "Pion";
	else if(pole_doc_typ % 10 == 2)
		return "Wieza";
	else if(pole_doc_typ % 10 == 3)
		return "Goniec";
	else if(pole_doc_typ % 10 == 4)
		return "Skoczek";
	else if(pole_doc_typ % 10 == 5)
		return "Hetman";
	else if(pole_doc_typ % 10 == 6)
		return "Krol";
	
	return "Puste pole";
}

void wybierz_rozpoczecie(int** plansza, int* pole_pocz, int* pole_doc, bool czyj_ruch) 
{
	int losuj = rand() % 4;

	if(czyj_ruch == 0) {
		switch(losuj) {
			case 0:
				pole_pocz[0] = 6;
				pole_pocz[1] = 4;
				pole_doc[0] = 4;
				pole_doc[1] = 4;
				break;
			case 1:
				pole_pocz[0] = 6;
				pole_pocz[1] = 3;
				pole_doc[0] = 4;
				pole_doc[1] = 3;
				break;
			case 2:
				pole_pocz[0] = 7;
				pole_pocz[1] = 6;
				pole_doc[0] = 5;
				pole_doc[1] = 5;
				break;
			case 3:
				pole_pocz[0] = 7;
				pole_pocz[1] = 1;
				pole_doc[0] = 5;
				pole_doc[1] = 2;
				break;
		}
	}
	else {
		switch(losuj) {
			case 0:
				pole_pocz[0] = 1;
				pole_pocz[1] = 4;
				pole_doc[0] = 3;
				pole_doc[1] = 4;
				break;
			case 1:
				pole_pocz[0] = 1;
				pole_pocz[1] = 3;
				pole_doc[0] = 3;
				pole_doc[1] = 3;
				break;
			case 2:
				pole_pocz[0] = 0;
				pole_pocz[1] = 6;
				pole_doc[0] = 2;
				pole_doc[1] = 5;
				break;
			case 3:
				pole_pocz[0] = 0;
				pole_pocz[1] = 1;
				pole_doc[0] = 2;
				pole_doc[1] = 2;
				break;
		}
	}
	
}


//GPU tworzenie grafu
void licz_graf_GPU(int* plansza, vector<int> &punkty, int depth, int max_depth, bool czyj_ruch, bool czy_wezel_niepusty)
{
	if(depth == max_depth) {
		int punkty_wezel = 0;
		int* h_punkty = new int[64];
		bool czyj_ruch_nowy = 0;
		if(depth % 2 == 0)
			czyj_ruch_nowy = czyj_ruch;
		else
			czyj_ruch_nowy = !czyj_ruch;

		if(czy_wezel_niepusty == 1) {
			int* h_ruchy = new int[64];
			int* d_punkty;
        	int* d_plansza;

			hipMalloc((void**)&d_plansza, 64 * sizeof(int));
			hipMalloc((void**)&d_punkty, 64 * sizeof(int));

			hipMemcpy(d_plansza, plansza, 64 * sizeof(int), hipMemcpyHostToDevice);
			kernelEvalPoints<<<1,64>>>(d_plansza, d_punkty, czyj_ruch_nowy);
			hipDeviceSynchronize();
			hipMemcpy(h_punkty, d_punkty, sizeof(int), hipMemcpyDeviceToHost);
			punkty_wezel = h_punkty[0];

			delete[] h_punkty;
			hipFree(d_punkty);
			hipFree(d_plansza);
		}
		else {
			if(max_depth % 2 == 1)
				punkty_wezel = MINIMAX_MINVAL;
			else
				punkty_wezel = MINIMAX_MAXVAL;
		}
		punkty.push_back(punkty_wezel);	
	}
	else if(czy_wezel_niepusty == 0) {
		for(int i = 0; i < 32; i++) {
			licz_graf_GPU(plansza, punkty, depth + 1, max_depth, !czyj_ruch, 0);
		}
	}
	else {
		//Znajdowanie pionkow
		int* h_ruchy = new int[64];
		int* d_ruchy;
        int* d_plansza;
		hipMalloc((void**)&d_plansza, 64 * sizeof(int));
		hipMalloc((void**)&d_ruchy, 64 * sizeof(int));

		hipMemcpy(d_plansza, plansza, 64 * sizeof(int), hipMemcpyHostToDevice);

		vector<int> pionki;
		kernelFindPawns<<<1,64>>>(d_plansza, d_ruchy, czyj_ruch);
		//Sortowanie
		thrust::device_ptr<int> thrust_tab(d_ruchy);
		thrust::sort(thrust_tab, thrust_tab + 64); 
		for(int i = 0; i < 16; i++) {
			if(thrust_tab[i] < 64) {
				pionki.push_back(thrust_tab[i]);
			}
			else
				break;
		}

		//Szukanie mozliwych ruchow
		vector<int> ruchy_start;
		vector<int> ruchy_doc;
		
		for(int i = 0; i < pionki.size(); i++) {
			if(ruchy_start.size() >= 32)
				break;

			kernelCheckAllMoves<<<1,64>>>(d_plansza, d_ruchy, pionki[i], czyj_ruch);
			hipDeviceSynchronize();
			//Sortowanie
			thrust::device_ptr<int> thrust_tab(d_ruchy);
			thrust::sort(thrust_tab, thrust_tab + 64); 
			for(int j = 0; j < 64; j++) {
				if(thrust_tab[j] < 64) {
					ruchy_start.push_back(pionki[i]);
					ruchy_doc.push_back(thrust_tab[j]);
				}
				else
					break;
			}
		}
		delete[] h_ruchy;
		hipFree(d_ruchy);
		hipFree(d_plansza);
		
		//Rozszerzanie grafu dla wyliczonych ruchow
		for(int i = 0; i < 32; i++) {
			if(i >= ruchy_start.size())
				licz_graf_GPU(plansza, punkty, depth + 1, max_depth, !czyj_ruch, 0);
			else{
				int start_id = ruchy_start[i];
				int doc_id = ruchy_doc[i];
				int pole_start_typ = plansza[start_id];
				int pole_doc_typ = plansza[doc_id];

				plansza[doc_id] = pole_start_typ;
				plansza[start_id] = 0;
				//Wybor przez bota hetmana w razie promocji
				if(pole_start_typ == 0 && (doc_id/8 == 0 || doc_id/8 == 7)) {
					int pole_prom = 0;
					if(czyj_ruch == 0)
						pole_prom = 5;
					else
						pole_prom = 15;
					plansza[doc_id] = pole_prom;
				}
					
				licz_graf_GPU(plansza, punkty, depth + 1, max_depth, !czyj_ruch, 1);
				plansza[doc_id] = pole_doc_typ;
				plansza[start_id] = pole_start_typ;
			}
		}
	}
}

void znajdz_najlepszy_ruch(int* plansza, vector<int> punkty, int* max_ruch, bool czyj_ruch)
{
	//Znajdowanie pionkow
	int* h_ruchy = new int[64];
	int* d_ruchy;
    int* d_plansza;
	hipMalloc((void**)&d_plansza, 64 * sizeof(int));
	hipMalloc((void**)&d_ruchy, 64 * sizeof(int));

	hipMemcpy(d_plansza, plansza, 64 * sizeof(int), hipMemcpyHostToDevice);

	vector<int> pionki;
	kernelFindPawns<<<1,64>>>(d_plansza, d_ruchy, czyj_ruch);
	//Sortowanie
	thrust::device_ptr<int> thrust_tab(d_ruchy);
	thrust::sort(thrust_tab, thrust_tab + 64); 
	for(int i = 0; i < 16; i++) {
		if(thrust_tab[i] < 64) {
			pionki.push_back(thrust_tab[i]);
		}
	else
		break;
	}

	//Szukanie mozliwych ruchow
	vector<int> ruchy_start;
	vector<int> ruchy_doc;
		
	for(int i = 0; i < pionki.size(); i++) {
		if(ruchy_start.size() >= 32)
			break;

		kernelCheckAllMoves<<<1,64>>>(d_plansza, d_ruchy, pionki[i], czyj_ruch);
		hipDeviceSynchronize();
		//Sortowanie
		thrust::device_ptr<int> thrust_tab(d_ruchy);
		thrust::sort(thrust_tab, thrust_tab + 64); 
		for(int j = 0; j < 64; j++) {
			if(thrust_tab[j] < 64) {
				ruchy_start.push_back(pionki[i]);
				ruchy_doc.push_back(thrust_tab[j]);
			}
			else
				break;
		}
	}
	delete[] h_ruchy;
	hipFree(d_ruchy);
	hipFree(d_plansza);

	int max_val = MINIMAX_MINVAL;
	int max_id_start = 0;
	int max_id_doc = 0;
	for(int i = 0; i < MAX_MOVES; i++) {
		if(punkty[i] >= max_val) {
			max_val = punkty[i];
			max_id_start = ruchy_start[i];
			max_id_doc = ruchy_doc[i];
		}
	}

	max_ruch[0] = max_id_start;
	max_ruch[1] = max_id_doc;
}



//--------------------------------------------Main------------------------------------------------------------------
int main()
{
	srand(time(NULL));
	//Tworzenie dwuwymiarowej tablicy 8x8 odpowiadajacej za figury znajdujace sie na polach
	int** plansza = new int* [8];
	int* gracze = new int[2];
	for (int i = 0; i < 8; i++) {
		plansza[i] = new int[8];
	}
	resetuj_plansze(plansza, gracze);
	bool czyj_ruch = 0;	//0 - biale, 1 - czarne
	int runda = 1;
	
	while(true) {
		system("clear");
		cout << "Wpisz 'r' w celu zresetowania gry" << endl << endl << endl;
		
		//Rysowanie planszy i calego "GUI"
		rysuj_plansze(plansza);
		if(czyj_ruch == 0)
			cout << "Obecny ruch - gracz BIALY" << endl << endl;
		else
			cout << "Obecny ruch - gracz CZARNY" << endl << endl;

		int* h_ruchy = new int[64];
		int* d_ruchy;
        int* d_plansza;

		hipMalloc((void**)&d_plansza, 64 * sizeof(int));
		hipMalloc((void**)&d_ruchy, 64 * sizeof(int));

		int* plansza_vec = new int[64];
		for(int i = 0; i < 64; i++) {
			plansza_vec[i] = plansza[i/8][i%8];
		}

		hipMemcpy(d_plansza, plansza_vec, 64 * sizeof(int), hipMemcpyHostToDevice);
		kernelEvalPoints<<<1, 64>>>(d_plansza, d_ruchy, czyj_ruch);
		hipDeviceSynchronize();
		hipMemcpy(h_ruchy, d_ruchy, sizeof(int), hipMemcpyDeviceToHost);

		//cout << "Obecna plansza punkty: " << h_ruchy[0] << endl;

		int* pole_poczatkowe = new int[2];
		int* pole_docelowe = new int[2];
		int pionek;

		//Runda gracza(człowieka)
		if(gracze[czyj_ruch] == 0) {
			//Pobieramy od gracza pole piona, ktory chce ruszyc i sprawdzamy,
			//czy gracz posiada pion na takim polu
			if(czyj_ruch == 0 || czyj_ruch == 1) {
				string pole_str = "";
				cout << "Podaj pole pionka:" << endl;
				cin >> pole_str;
				pole_poczatkowe = new int[2];
				if(tolower(pole_str[0]) == 'r') {
					cout << "Gra zostanie zresetowana!" << endl;
					system("pause");
					resetuj_plansze(plansza, gracze);
					czyj_ruch = 0;
					runda = 1;
					continue;
				}
				if(string_na_pole(plansza, pole_str, pole_poczatkowe) == 0) {
					cout << "Nieprawidlowe pole!" << endl;
					system("pause");
					continue;
				}
				
				pionek = sprawdz_pionek(plansza, pole_poczatkowe, czyj_ruch);
				if(pionek == 0) {
					cout << "Nie masz pionka na tym polu!" << endl;
					system("pause");
					continue;
				}

				int start = pole_poczatkowe[0] * 8  + pole_poczatkowe[1];
				hipMemcpy(d_plansza, plansza_vec, 64 * sizeof(int), hipMemcpyHostToDevice);
				kernelCheckAllMoves<<<1, 64>>>(d_plansza, d_ruchy, start, czyj_ruch);
				hipDeviceSynchronize();
				hipMemcpy(h_ruchy, d_ruchy, 64 * sizeof(int), hipMemcpyDeviceToHost);

				vector<int> pionek_ruchy;
				for(int i = 0; i < 64; i++) {
					if(h_ruchy[i] < 64)
						pionek_ruchy.push_back(h_ruchy[i]);
				}

				if(pionek_ruchy.empty()) {
					cout << "Pionek ten nie moze wykonac zadnego ruchu!" << endl;
					system("pause");
					continue;
				}
				else {
					cout << "Mozliwe ruchy:" << endl;
					for(int i = 0; i < pionek_ruchy.size(); i++) {
						int* pole_temp = new int[2];
						pole_temp[0] = pionek_ruchy[i] / 8;
						pole_temp[1] = pionek_ruchy[i] % 8;
						string temp_str = pole_na_string(plansza, pole_temp);
						cout << temp_str[0] << temp_str[1] << " ";
						if(i % 5 == 0 && i != 0)
							cout << endl;
						delete []pole_temp;
					}
					cout << endl;
				}
		
				
				//Pobieramy od gracza pole, na ktore chce ruszyc sie poprzednio wybranym pionem
				//i sprawdzamy, czy ruch jest prawidlowy
				cout << "Podaj ruch pionka:" << endl;
				cin >> pole_str;
				pole_docelowe = new int[2];
				if(tolower(pole_str[0]) == 'r') {
					cout << "Gra zostanie zresetowana!" << endl;
					system("pause");
					resetuj_plansze(plansza, gracze);
					czyj_ruch = 0;
					runda = 1;
					continue;
				}
				else if(string_na_pole(plansza, pole_str, pole_docelowe) == 0) {
					cout << "Nieprawidlowe pole!" << endl;
					system("pause");
					continue;
				}
				else if(pole_docelowe[0] == pole_poczatkowe[0] && pole_docelowe[1] == pole_poczatkowe[1]) {
					cout << "Nieprawidlowe pole!" << endl;
					system("pause");
					continue;
				}

				hipMemcpy(d_plansza, plansza_vec, 64 * sizeof(int), hipMemcpyHostToDevice);
				start = pole_poczatkowe[0] * 8  + pole_poczatkowe[1];
				int koniec = pole_docelowe[0] * 8  + pole_docelowe[1];
				kernelCheckMove<<<1, 1>>>(d_plansza, d_ruchy, start, koniec, czyj_ruch);
				hipDeviceSynchronize();
				hipMemcpy(h_ruchy, d_ruchy, sizeof(int), hipMemcpyDeviceToHost);

				if(h_ruchy[0] > 64) {
					cout << "Nie mozesz wykonac takiego ruchu!" << endl;
					system("pause");
					continue;
				}
			}
			
			//Sprawdzanie promocji piona, wybor figury przez gracza jesli nastapila ona
			if(pionek == 1) {
				bool promocja = 0;
				if(czyj_ruch == 0 && pole_docelowe[0] == 0)
					promocja = 1;
				else if(czyj_ruch == 1 && pole_docelowe[0] == 7)
					promocja = 1;
				
				if(promocja == 1) {
					int figura_promocja = 0;
					cout << "Nastapila promocja! Wybierz figure, do ktorej ma awansowac pion:" << endl;
					cout << "2 - wieza" << endl;
					cout << "3 - goniec" << endl;
					cout << "4 - skoczek" << endl;
					cout << "5 - hetman" << endl;
					cin >> figura_promocja;
					if(figura_promocja >= 2 && figura_promocja <= 5) {
						pionek = figura_promocja;
					}	
					else {
						cout << "Nieprawidlowo wybrana figura!" << endl;
						system("pause");
						continue;
					}
				}
			}
		}
		//Pierwsza runda AI - wybor pierwszego ruchu
		else if(runda == 1 || runda == 2) {
			wybierz_rozpoczecie(plansza, pole_poczatkowe, pole_docelowe, czyj_ruch);
			pionek = plansza[pole_poczatkowe[0]][pole_poczatkowe[1]];

			string str_pocz = pole_na_string(plansza, pole_poczatkowe);
			string str_doc = pole_na_string(plansza, pole_docelowe);
			cout << "Wybrany ruch to " << str_pocz[0] << str_pocz[1] << "->" <<
				str_doc[0] << str_doc[1] << endl;
			
			string pole_str;
			cout << "Wpisz 'r' aby zrestartowac rozgrywke, lub cokolwiek innego, żeby AI wykonało ruch:" << endl;
			cin >> pole_str;
			if(tolower(pole_str[0]) == 'r') {
				cout << "Gra zostanie zresetowana!" << endl;
				system("pause");
				resetuj_plansze(plansza, gracze);
				czyj_ruch = 0;
				runda = 1;
				continue;
			}
		}
		//Runda AI
		else {
			vector<int> punkty_minimax;
			int max_glebokosc = 3;
			unsigned int rozmiar_minimax = pow(MAX_MOVES, max_glebokosc);
			licz_graf_GPU(plansza_vec, punkty_minimax, 0, max_glebokosc, czyj_ruch, 1);
			int* host_points = new int[punkty_minimax.size()];
			for(int i = 0; i < punkty_minimax.size(); i++) {
				host_points[i] = punkty_minimax[i];
			}

			int* dev_points;
			int* dev_max;
			int* dev_min;
			
			//Alokacja pamieci
			hipMalloc((void**)&dev_points, rozmiar_minimax * sizeof(int));
			hipMalloc((void**)&dev_max, rozmiar_minimax * sizeof(int));
			hipMalloc((void**)&dev_min, rozmiar_minimax * sizeof(int));

			hipMemcpy(dev_points, host_points, rozmiar_minimax * sizeof(int), hipMemcpyHostToDevice);

			int watki = MAX_MOVES;
			unsigned int bloki = 1024;
			unsigned int max_blocks = pow(2, 31) - 1;

			for(int i = max_glebokosc; i > 1; i--) {
				unsigned int rozmiar_wynik = pow(MAX_MOVES, i-1);
				bloki = min(max_blocks, rozmiar_wynik);
				if(i % 2 == 0) {
					if(i == max_glebokosc) {
						kernelMin<<<bloki, watki>>>(dev_min, dev_points, rozmiar_wynik);
						hipDeviceSynchronize();
					}	
					else {
						kernelMin<<<bloki, watki>>>(dev_min, dev_max, rozmiar_wynik);
						hipDeviceSynchronize();
					}
				}
				else {
					if(i == max_glebokosc) {
						kernelMax<<<bloki, watki>>>(dev_max, dev_points, rozmiar_wynik);
						hipDeviceSynchronize();
					}	
					else {
						kernelMax<<<bloki, watki>>>(dev_max, dev_min, rozmiar_wynik);
						hipDeviceSynchronize();
					}	
				}
			}

			
			vector<int> punkty_ostatni_wezel;
			if(max_glebokosc != 1) {
				hipMemcpy(host_points, dev_min, MAX_MOVES * sizeof(int), hipMemcpyDeviceToHost);
				for(int i = 0; i < MAX_MOVES; i++) {
					punkty_ostatni_wezel.push_back(host_points[i]);
					//cout << punkty_ostatni_wezel[i] << " ";
				}
			}
			else {
				for(int i = 0; i < MAX_MOVES; i++) {
					punkty_ostatni_wezel.push_back(punkty_minimax[i]);
					//cout << punkty_ostatni_wezel[i] << " ";
				}
			}

			int* najlepszy_ruch = new int[2];
			znajdz_najlepszy_ruch(plansza_vec, punkty_ostatni_wezel, najlepszy_ruch, czyj_ruch);
			pole_poczatkowe[0] = najlepszy_ruch[0] / 8;
			pole_poczatkowe[1] = najlepszy_ruch[0] % 8;
			pole_docelowe[0] = najlepszy_ruch[1] / 8;
			pole_docelowe[1] = najlepszy_ruch[1] % 8;
			pionek = plansza[pole_poczatkowe[0]][pole_poczatkowe[1]] % 10;
			//Promocja
			if(pionek == 1 && (pole_docelowe[0] == 0 || pole_docelowe[0] == 7))	
				pionek = 5;	

			string str_pocz = pole_na_string(plansza, pole_poczatkowe);
			string str_doc = pole_na_string(plansza, pole_docelowe);
			cout << "Najlepszy wykryty ruch to " << str_pocz[0] << str_pocz[1] << "->" <<
				str_doc[0] << str_doc[1] << endl;
			
			string pole_str;
			cout << "Wpisz 'r' aby zrestartowac rozgrywke, lub cokolwiek innego, żeby AI wykonało ruch:" << endl;
			cin >> pole_str;
			if(tolower(pole_str[0]) == 'r') {
				cout << "Gra zostanie zresetowana!" << endl;
				system("pause");
				resetuj_plansze(plansza, gracze);
				czyj_ruch = 0;
				runda = 1;
				continue;
			}

			delete[] najlepszy_ruch;
			hipFree(dev_points);
			hipFree(dev_min);
			hipFree(dev_max);
		}	


		//Sprawdzanie szach mat, ewentualne resetowanie gry
		if(plansza[pole_docelowe[0]][pole_docelowe[1]] % 10 == 6) {
			cout << "=================================" << endl;
			if(czyj_ruch == 0)
				cout << "Koniec gry! Wygrywa gracz bialy!" << endl;
			else
				cout << "Koniec gry! Wygrywa gracz Czarny!" << endl;
			cout << "=================================" << endl;
			string str;
			cin >> str;
			system("pause");
			resetuj_plansze(plansza, gracze);
			czyj_ruch = 0;
			runda = 1;
			continue;
		}

		//Przesuwanie pionka na nowe pole
		if(czyj_ruch == 1)
			pionek += 10;
		plansza[pole_poczatkowe[0]][pole_poczatkowe[1]] = 0;
		plansza[pole_docelowe[0]][pole_docelowe[1]] = pionek;
		
		//Zmiana grajacego gracza
		czyj_ruch = !czyj_ruch;
		runda++;

		delete []pole_poczatkowe;
		delete []pole_docelowe;
		hipFree(d_plansza);
		hipFree(d_ruchy);
	}

	delete []gracze;
	return 0;
}